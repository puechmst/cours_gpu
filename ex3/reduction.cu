#include<hip/hip_runtime.h>
#include<iostream>



#define BLOCK_DIM (256)

__global__ void reduction(float *a, float *b, int n) {
    __shared__ float r[BLOCK_DIM];
    int i = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int nb = (n + BLOCK_DIM -1) / BLOCK_DIM;
    int step = BLOCK_DIM >> 1;
    if (i < n) 
        r[threadIdx.x] = a[i];
    else 
        r[threadIdx.x] = 0.0f;
    __syncthreads();
    while(step >= 1) {
        if(threadIdx.x < step)
           r[threadIdx.x] += r[threadIdx.x + step];
        __syncthreads();
        step >>= 1;
    }
    if(threadIdx.x == 0)
        b[blockIdx.x] = r[0];
    __syncthreads();
    if(i == 0 && nb > 1) 
            reduction<<<nb, BLOCK_DIM,0,cudaStreamTailLaunch>>>(b,b,nb);
}

float device_reduction(float *a, int n) {
    float *da;
    float *dbuffer;
    float hbuffer;
    float res;
    int sz = (n + BLOCK_DIM - 1) / BLOCK_DIM;
    hipMalloc(&da, n * sizeof(float));
    hipMalloc(&dbuffer, sz * sizeof(float));
    hipMemcpy(da, a, n * sizeof(float),hipMemcpyHostToDevice);
    reduction<<<sz, BLOCK_DIM>>>(da, dbuffer, n);
    hipDeviceSynchronize();
    hipMemcpy(&hbuffer, dbuffer, sizeof(float),hipMemcpyDeviceToHost);
    res = hbuffer;
    hipFree(da);
    hipFree(dbuffer);
    return res;
}