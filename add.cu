#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<random>
#include<iostream>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include "add.h"
#define THREADS_PER_BLOCK (512)

__global__ void add_kernel(int n, float *u, float *v, float *w) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        w[i] = u[i] + v[i];
    }
}

__global__ void setup_rnd_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
 
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void add_init_kernel(hiprandState *state,
                                int n,
                                float *u)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        /* Copy state to local memory for efficiency */
        hiprandState localState = state[i];
        u[i] = hiprand_uniform(&localState); 
        /* Copy state back to global memory */
        state[i] = localState;
    }
}

void rnd_init(int n, float *u) {
    // for debugging purpose, seed has a fixed value
    std::mt19937 gen(5);

    for (int i = 0 ; i < n ; i++)
    // random generator from c++ standard library
        u[i] = gen();
}

hipError_t alloc_debug(void **ptr, size_t sz) {
  hipError_t err;
    err = hipMalloc(ptr, sz);
    switch(err) {
        case hipErrorOutOfMemory:
            fprintf(stderr, "Cannot allocated memory on device.\n");
            break;
        case hipErrorInvalidValue:
            fprintf(stderr, "Invalid value in arguments.\n");
            break;
    }
    return err;
}

hipError_t alloc_managed_debug(void ** ptr, size_t sz) {
    hipError_t err;
    err = hipMallocManaged(ptr, sz);
    switch(err) {
        case hipErrorOutOfMemory:
            fprintf(stderr, "Cannot allocated memory on device.\n");
            break;
        case hipErrorNotSupported:
            fprintf(stderr, "Managed memory is not available.\n");
            break;
        case hipErrorInvalidValue:
            fprintf(stderr, "Invalid value in arguments.\n");
            break;
    }
    return err;
}

int add_unified(int n) {
    float *a, *b, *c;
    hiprandState *rnd_states;
    int sz;
    int n_blocks;
    int b_pass;
    sz = n * sizeof(float);
    if(alloc_managed_debug((void **)&a, sz) != hipSuccess) return 0;
    if(alloc_managed_debug((void **)&b, sz) != hipSuccess) return 0;
    if(alloc_managed_debug((void **)&c, sz) != hipSuccess) return 0;
    if(alloc_managed_debug((void **)&rnd_states, n * sizeof(hiprandState)) != hipSuccess) return 0;

    n_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    std::cout << n_blocks * THREADS_PER_BLOCK << " threads allocated for size " << n << std::endl;
    //rnd_init(n, a);
    //rnd_init(n, b);
    setup_rnd_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states );
    hipDeviceSynchronize();
    add_init_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states, n, a );
    hipDeviceSynchronize();
    add_init_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states, n, b );
    hipDeviceSynchronize();
    add_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, a, b, c);
    hipDeviceSynchronize();
     // verification
    b_pass = 1;
    for(int i = 0 ; i < n ; i++) {
       if (c[i] != a[i] + b[i]) {
            b_pass = 0;
            break;
       }
    }
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(rnd_states);
    return b_pass;
}

int  launch_and_test(int n) {
    float *a,*b,*c;
    float *da,*db,*dc;
    int n_blocks;
    int b_pass;
    int sz;

    sz = n * sizeof(float);
    a = (float *)malloc(sz);
    b = (float *)malloc(sz);
    c = (float *)malloc(sz);

    if(alloc_debug((void **)&da, sz) != hipSuccess) return 0;
    if(alloc_debug((void **)&db, sz) != hipSuccess) return 0;
    if(alloc_debug((void **)&dc, sz) != hipSuccess) return 0;

    // remplissage aleatoire
    rnd_init(n, a); 
    rnd_init(n, b);

    hipMemcpy(da, a, sz, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sz, hipMemcpyHostToDevice);

    n_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    std::cout << n_blocks * THREADS_PER_BLOCK << " threads allocated for size " << n << std::endl;
    add_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, da, db, dc);
    hipDeviceSynchronize();
    hipMemcpy(c, dc, sz, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // verification
    b_pass = 1;
    for(int i = 0 ; i < n ; i++) {
       if (c[i] != a[i] + b[i]) {
            b_pass = 0;
            break;
       }
    }

    free(a);
    free(b);
    free(c);

    return b_pass;
}