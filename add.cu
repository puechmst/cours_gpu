#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand_kernel.h>
#include "add.h"
#define THREADS_PER_BLOCK (512)

__global__ void add_kernel(int n, float *u, float *v, float *w) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        w[i] = u[i] + v[i];
    }
}

__global__ void setup_rnd_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
 
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void add_init_kernel(hiprandState *state,
                                int n,
                                float *u)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        /* Copy state to local memory for efficiency */
        hiprandState localState = state[i];
        u[i] = hiprand_uniform(&localState); 
        /* Copy state back to global memory */
        state[i] = localState;
    }
}

void rnd_init(int n, float *u) {
    for (int i = 0 ; i < n ; i++)
        u[i] = drand48();
}

int add_unified(int n) {
    float *a, *b, *c;
    hiprandState *rnd_states;
    int sz;
    int n_blocks;
    int b_pass;
    sz = n * sizeof(float);
    hipMallocManaged((void **)&a, sz);
    hipMallocManaged((void **)&b, sz);
    hipMallocManaged((void **)&c, sz);
    hipMallocManaged((void **)&rnd_states, n * sizeof(hiprandState));

    n_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    //rnd_init(n, a);
    //rnd_init(n, b);
    setup_rnd_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states );
    hipDeviceSynchronize();
    add_init_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states, n, a );
    hipDeviceSynchronize();
    add_init_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(rnd_states, n, b );
    hipDeviceSynchronize();
    add_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, a, b, c);
    hipDeviceSynchronize();
     // verification
    b_pass = 1;
    for(int i = 0 ; i < n ; i++) {
       if (c[i] != a[i] + b[i]) {
            b_pass = 0;
            break;
       }
    }
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(rnd_states);
    return b_pass;
}

int  launch_and_test(int n) {
    float *a,*b,*c;
    float *da,*db,*dc;
    int n_blocks;
    int b_pass;
    int sz;

    sz = n * sizeof(float);
    a = (float *)malloc(sz);
    b = (float *)malloc(sz);
    c = (float *)malloc(sz);

    hipMalloc((void **)&da, sz);
    hipMalloc((void **)&db, sz);
    hipMalloc((void **)&dc, sz);

    // remplissage aleatoire
    rnd_init(n, a); 
    rnd_init(n, b);

    hipMemcpy(da, a, sz, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sz, hipMemcpyHostToDevice);

    n_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_kernel<<<n_blocks, THREADS_PER_BLOCK>>>(n, da, db, dc);
    hipDeviceSynchronize();
    hipMemcpy(c, dc, sz, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // verification
    b_pass = 1;
    for(int i = 0 ; i < n ; i++) {
       if (c[i] != a[i] + b[i]) {
            b_pass = 0;
            break;
       }
    }

    free(a);
    free(b);
    free(c);

    return b_pass;
}