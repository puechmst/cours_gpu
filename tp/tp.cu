#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#define SIZE (100)
#define HSIZE (10)
#define NROW (100)
#define NCOL (100)
#define NRH (10)
#define NCH (10)
#define BSIZE (64)
#define BX (8)
#define BY (8)

void cpu_conv1d(int n, float *x, int p, float *h, float *y)
{
   // calcul du produit de convolution 1d sur CPU.
   for (int i = 0; i < n; i++)
   {
      float s = 0.0;
      for (int j = max(0, i - p + 1); j <= i; j++)
      {
         s += x[j] * h[i - j];
      }
      y[i] = s;
   }
}

void cpu_conv2d(int m, int n, float *x, int p, int q, float *h, float *y)
{
   // calcul du produit de convolution 2D sur CPU.
   for (int i = 0; i < m; i++)
   {
      for (int j = 0; j < n; j++)
      {
         float s = 0.0;
         for (int k = max(0, i - p + 1); k <= i; k++)
         {
            for (int l = max(0, j - q + 1); l <= j; l++)
               s += x[k * n + l] * h[(i - k) * q + j - l];
         }
         y[i * n + j] = s;
      }
   }
}

__global__ void gpu_conv1d(int n, float *x, int p, float *h, float *y)
{
   int i = threadIdx.x + blockIdx.x * BSIZE;
   if (i < n)
   {
      float s = 0.0;
      for (int j = max(0, i - HSIZE + 1); j <= i; j++)
      {
         s += x[j] * h[i - j];
      }
      y[i] = s;
   }
} 

__global__ void gpu_conv2d(int m, int n, float *x, int p, int q, float *h, float *y)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;

   if (i < m && j < n)
    {
      float s = 0.0;
      for (int k = max(0, i - p + 1); k <= i; k++)
         {
            for (int l = max(0, j - q + 1); l <= j; l++)
               s += x[k * n + l] * h[(i - k) * q + j - l];
         }
      y[i * n + j] = s;
   }
} 

float test_conv1d() {
   float *x, *y, *h;
   float *dx, *dy, *dh;
   int nb;

   x = new float[SIZE];
   y = new float[SIZE];
   h = new float[HSIZE];

   for (int i = 0; i < SIZE; i++)
      x[i] = 1.0f;

   for (int i = 0; i < HSIZE; i++)
      h[i] = 1.0f;

   nb = (SIZE + BSIZE - 1) / BSIZE;
   hipMalloc(&dx, SIZE * sizeof(float));
   hipMalloc(&dy, SIZE * sizeof(float));
   hipMalloc(&dh, HSIZE * sizeof(float));
   hipMemcpy(dx, x, SIZE * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(dh, h, HSIZE * sizeof(float), hipMemcpyHostToDevice);
   cpu_conv1d(SIZE, x, HSIZE, h, y);
   gpu_conv1d<<<nb, BSIZE>>>(SIZE, dx, HSIZE, dh, dy);
   hipMemcpy(x, dy, SIZE * sizeof(float), hipMemcpyDeviceToHost);
   float err = 0.0f;
   for (int i = 0; i < SIZE; i++)
      err += fabsf(y[i] - x[i]);
   hipFree(dx);
   hipFree(dy);
   hipFree(dh);
   delete[] x;
   delete[] y;
   delete[] h;
   return err;
}

float test_conv2d() {
   float *x, *y, *h;
   float *dx, *dy, *dh;
   int nbx, nby;

   x = new float[NROW * NCOL];
   y = new float[NROW * NCOL];
   h = new float[NRH * NCH];

   for (int i = 0; i < NROW * NCOL; i++)
      x[i] = 1.0f;

   for (int i = 0; i < NRH * NCH; i++)
      h[i] = 1.0f;

   nbx = (NROW + BX - 1) / BX;
   nby = (NCOL + BY - 1) / BY;
   hipMalloc(&dx, NROW * NCOL * sizeof(float));
   hipMalloc(&dy, NROW * NCOL * sizeof(float));
   hipMalloc(&dh, NRH * NCH * sizeof(float));
   hipMemcpy(dx, x, NROW * NCOL * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(dh, h, NRH * NCH * sizeof(float), hipMemcpyHostToDevice);
   cpu_conv2d(NROW, NCOL, x, NRH, NCH, h, y);
   gpu_conv2d<<<dim3(nbx, nby, 1), dim3(BX, BY, 1)>>>(NROW, NCOL, dx, NRH, NCH, dh, dy);
   hipMemcpy(x, dy, NROW * NCOL * sizeof(float), hipMemcpyDeviceToHost);
   float err = 0.0f;
   for (int i = 0; i < NROW * NCOL; i++)
      err += fabsf(y[i] - x[i]);
   hipFree(dx);
   hipFree(dy);
   hipFree(dh);
   delete[] x;
   delete[] y;
   delete[] h;
   return err;
}


int main(int argc, char *argv[])
{
   std::cout << test_conv1d() << '\t' << test_conv2d()<< std::endl;
   return 0;
}